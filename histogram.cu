#include "hip/hip_runtime.h"
#include "readTextFile.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N_LETTERS 26

void seqentialHistogram(unsigned char *data, int length, unsigned int *histo, int nBins)
{
	for (int i = 0; i < length; i++)
	{
		int alphabetPosition = data[i] - 'a';
		if (alphabetPosition >= 0 && alphabetPosition < N_LETTERS)
		{
			histo[(alphabetPosition*nBins) / N_LETTERS]++;
		}
	}
}

// Histogram - basic parallel implementation
__global__ void histogram_1(unsigned char *buffer, long size, unsigned int *histogram, unsigned int nBins)
{
	long index = blockIdx.x * blockDim.x + threadIdx.x;
	long threadSize = ((size - 1) / (gridDim.x * blockDim.x)) + 1;

	long startPos = index * threadSize;
	for (long i = startPos; i < startPos + threadSize && i < size; i++)
	{
		int pos = (int)buffer[i];
		pos -= 'a';
		if (pos > -1 && pos < N_LETTERS)
			atomicAdd(&(histogram[(pos*nBins) / N_LETTERS]), 1);
	}
}

// Histogram - interleaved partitioning
__global__ void histogram_2(unsigned char *buffer, long size, unsigned int *histogram, unsigned int nBins)
{
	long index = blockIdx.x * blockDim.x + threadIdx.x;
	long threadsCount = (gridDim.x * blockDim.x);

	if (index < size)
		for (long i = index; i < size; i += threadsCount)
		{
			int pos = (int)buffer[i];
			pos -= 'a';
			if (pos > -1 && pos < N_LETTERS)
				atomicAdd(&(histogram[(pos*nBins) / N_LETTERS]), 1);
		}
}

// Histogram - interleaved partitioning + privatisation
__global__ void histogram_3(unsigned char *buffer, long size, unsigned int *histogram, unsigned int nBins)
{
	extern __shared__ unsigned int s_histogram[];

	for (int i = threadIdx.x; i < nBins; i += blockDim.x)
		s_histogram[i] = 0;

	__syncthreads();

	long blockSize = ((size - 1) / gridDim.x) + 1;
	long blockStart = blockIdx.x * blockSize;
	for (int i = blockStart + threadIdx.x; i < blockStart + blockSize && i < size; i += blockDim.x)
	{
		register int pos = (int)buffer[i];
		pos -= 'a';
		if (pos > -1 && pos < N_LETTERS)
			atomicAdd(&(s_histogram[(pos*nBins) / N_LETTERS]), 1);
	}

	__syncthreads();

	for (int i = threadIdx.x; i < nBins; i += blockDim.x)
		atomicAdd(&(histogram[i]), s_histogram[i]);
}

// Extra: Histogram - interleaved partitioning + privatisation + aggregation
__global__ void histogram_4(unsigned char *buffer, long size, unsigned int *histogram, unsigned int nBins)
{
	extern __shared__ unsigned int s_histogram[];

	for (int i = threadIdx.x; i < nBins; i += blockDim.x)
		s_histogram[i] = 0;

	__syncthreads();

	long blockSize = ((size - 1) / gridDim.x) + 1;
	long blockStart = blockIdx.x * blockSize;

	register int same_letters_counter = 0;
	int last_letter = -9999;
	for (int i = blockStart + threadIdx.x; i < blockStart + blockSize && i < size; i += blockDim.x)
	{
		register int pos = (int)buffer[i];
		pos -= 'a';
		if (pos == last_letter)
			same_letters_counter++;
		else
		{
			if (last_letter > -1 && last_letter < N_LETTERS)
				atomicAdd(&(s_histogram[(last_letter * nBins) / N_LETTERS]), same_letters_counter);
			
			same_letters_counter=1;
			last_letter=pos;
		}
	}

	if (last_letter > -1 && last_letter < N_LETTERS)
		atomicAdd(&(s_histogram[(last_letter * nBins) / N_LETTERS]), same_letters_counter);

	__syncthreads();

	for (int i = threadIdx.x; i < nBins; i += blockDim.x)
		atomicAdd(&(histogram[i]), s_histogram[i]);
}

int main(int argc, char **argv)
{
	// check if number of input args is correct: input text filename
	if (argc < 2 || argc > 3)
	{
		printf("Wrong number of arguments! Expecting 1 mandatory argument (input .txt filename) and 1 optional argument (number of bins). \n");
		return 0;
	}

	// read input string
	long size = getNoChars(argv[1]) + 1;
	unsigned char *h_buffer = (unsigned char *)malloc(size * sizeof(unsigned char));
	readFile(argv[1], size, h_buffer);
	printf("Input string size: %ld\n", size);

	// set number of bins
	int nBins = 26;
	if (argc == 3)
	{
		int inBinsVal = atoi(argv[2]);
		if (inBinsVal <= N_LETTERS)
		{
			nBins = inBinsVal;
		}
	}

	// histograms init
	unsigned int *histogram1 = (unsigned int *)malloc(nBins * sizeof(unsigned int));
	unsigned int *histogram2 = (unsigned int *)malloc(nBins * sizeof(unsigned int));
	unsigned int *histogram3 = (unsigned int *)malloc(nBins * sizeof(unsigned int));
	unsigned int *histogram4 = (unsigned int *)malloc(nBins * sizeof(unsigned int));

	// cuda alloc
	unsigned char *d_buffer;
	unsigned int *distogram1;
	unsigned int *distogram2;
	unsigned int *distogram3;
	unsigned int *distogram4;

	hipMalloc((void **)&d_buffer, size * sizeof(unsigned char));
	hipMalloc((void **)&distogram1, nBins * sizeof(unsigned int));
	hipMalloc((void **)&distogram2, nBins * sizeof(unsigned int));
	hipMalloc((void **)&distogram3, nBins * sizeof(unsigned int));
	hipMalloc((void **)&distogram4, nBins * sizeof(unsigned int));

	// cuda run
	hipMemcpy(d_buffer, h_buffer, size * sizeof(unsigned char), hipMemcpyHostToDevice);

	dim3 dimGrid1d(60);
	dim3 dimBlock1d(256, 1, 1);

	histogram_1<<<dimGrid1d, dimBlock1d>>>(d_buffer, size, distogram1, nBins);
	histogram_2<<<dimGrid1d, dimBlock1d>>>(d_buffer, size, distogram2, nBins);
	histogram_3<<<dimGrid1d, dimBlock1d, nBins>>>(d_buffer, size, distogram3, nBins);
	histogram_4<<<dimGrid1d, dimBlock1d, nBins>>>(d_buffer, size, distogram4, nBins);

	hipMemcpy(histogram1, distogram1, nBins * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(histogram2, distogram2, nBins * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(histogram3, distogram3, nBins * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(histogram4, distogram4, nBins * sizeof(unsigned int), hipMemcpyDeviceToHost);

	// sequential
	unsigned int *histogram = (unsigned int *)malloc(nBins * sizeof(unsigned int));
	for (int i = 0; i < nBins; i++)
		histogram[i] = 0;

	seqentialHistogram(h_buffer, size, histogram, nBins);

	// printing
	printf("Histogram 0: ");
	for (int i = 0; i < nBins; i++)
		printf("%d ", histogram[i]);
	printf("\n");

	printf("Histogram 1: ");
	for (int i = 0; i < nBins; i++)
		printf("%d ", histogram1[i]);
	printf("\n");

	printf("Histogram 2: ");
	for (int i = 0; i < nBins; i++)
		printf("%d ", histogram2[i]);
	printf("\n");

	printf("Histogram 3: ");
	for (int i = 0; i < nBins; i++)
		printf("%d ", histogram3[i]);
	printf("\n");

	printf("Histogram 4: ");
	for (int i = 0; i < nBins; i++)
		printf("%d ", histogram4[i]);
	printf("\n");

	// free memory
	free(histogram);
	free(histogram1);
	free(histogram2);
	free(histogram3);
	free(histogram4);

	hipFree(d_buffer);
	hipFree(distogram1);
	hipFree(distogram2);
	hipFree(distogram3);
	hipFree(distogram4);

	///////////////////////////////////////////////////////

	free(h_buffer);

	// For error detection you can use the following code (don't forget to include iostream)
	// hipError_t err = hipGetLastError();
	// if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

	return 0;
}
